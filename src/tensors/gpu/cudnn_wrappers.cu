#include "hip/hip_runtime.h"
#include "tensors/gpu/cudnn_wrappers.h"

namespace marian {

#ifdef CUDNN

#include <hipDNN.h>

#define CUDNN_CALL(x)                 \
  do {                                \
    if((x) != HIPDNN_STATUS_SUCCESS) { \
      printf("Error (%s) at %s:%d\n", \
             hipdnnGetErrorString(x),  \
             __FILE__,                \
             __LINE__);               \
    }                                 \
  } while(0)

CUDNNWrapper::CUDNNWrapper() {
  CUDNN_CALL(hipdnnCreate(&cudnnHandle_));
}

CUDNNWrapper::~CUDNNWrapper() {
  // std::cerr << "destroy wrapper" << std::endl;
  CUDNN_CALL(hipdnnDestroy(cudnnHandle_));
}

void CUDNNWrapper::setCudnnTensor(hipdnnTensorDescriptor_t& desc, Tensor x) {
  setCudnnTensor(desc, x->shape());
}

void CUDNNWrapper::setCudnnTensor(hipdnnTensorDescriptor_t& desc,
                                  const Shape& shape) {
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        shape[0],
                                        shape[1],
                                        shape[2],
                                        shape[3]));
}

/******************************************************************************
 * ConvolutionWrapper
 *****************************************************************************/

ConvolutionWrapper::ConvolutionWrapper(const Shape& kernelShape,
                                       const Shape& biasShape,
                                       int hPad,
                                       int wPad,
                                       int hStride,
                                       int wStride) {
  setKernelDescriptor(kernelShape);
  setConvDescriptor(hPad, wPad, hStride, wStride);
  setCudnnTensor(biasDesc_, biasShape);
}

void ConvolutionWrapper::getOutputShape(const Shape& xShape, Shape& shape) {
  hipdnnTensorDescriptor_t xDesc;
  setCudnnTensor(xDesc, xShape);
  shape.resize(4);
  CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(convDesc_,
                                                   xDesc,
                                                   kernelDesc_,
                                                   shape.data(),
                                                   shape.data() + 1,
                                                   shape.data() + 2,
                                                   shape.data() + 3));
  hipdnnDestroyTensorDescriptor(xDesc);
}

void ConvolutionWrapper::forward(Tensor x,
                                 Tensor kernels,
                                 Tensor bias,
                                 Tensor y) {
  hipSetDevice(x->getDeviceId().no);

  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, x);
  setCudnnTensor(yDesc, y);

  const float alpha = 1.0f;
  const float beta = 0.0f;

  CUDNN_CALL(hipdnnConvolutionForward(cudnnHandle_,
                                     &alpha,
                                     xDesc,
                                     x->data(),
                                     kernelDesc_,
                                     kernels->data(),
                                     convDesc_,
                                     HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                     nullptr,
                                     0,
                                     &beta,
                                     yDesc,
                                     y->data()));
  CUDNN_CALL(hipdnnAddTensor(
      cudnnHandle_, &alpha, biasDesc_, bias->data(), &alpha, yDesc, y->data()));
  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

void ConvolutionWrapper::backward(Tensor x,
                                  Tensor xGrad,
                                  Tensor kernels,
                                  Tensor kernelGrad,
                                  Tensor biasGrad,
                                  Tensor yGrad) {
  hipSetDevice(xGrad->getDeviceId().no);

  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, xGrad);
  setCudnnTensor(yDesc, yGrad);

  const float alpha = 1.0f;
  const float beta = 1.0f;

  CUDNN_CALL(hipdnnConvolutionBackwardData(cudnnHandle_,
                                          &alpha,
                                          kernelDesc_,
                                          kernels->data(),
                                          yDesc,
                                          yGrad->data(),
                                          convDesc_,
                                          HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                                          nullptr,
                                          0,
                                          &beta,
                                          xDesc,
                                          xGrad->data()));

  CUDNN_CALL(hipdnnConvolutionBackwardFilter(cudnnHandle_,
                                            &alpha,
                                            xDesc,
                                            x->data(),
                                            yDesc,
                                            yGrad->data(),
                                            convDesc_,
                                            HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                            nullptr,
                                            0,
                                            &beta,
                                            kernelDesc_,
                                            kernelGrad->data()));

  CUDNN_CALL(hipdnnConvolutionBackwardBias(cudnnHandle_,
                                          &alpha,
                                          yDesc,
                                          yGrad->data(),
                                          &beta,
                                          biasDesc_,
                                          biasGrad->data()));

  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

ConvolutionWrapper::~ConvolutionWrapper() {
  // std::cerr << "destroy conv-wrapper" << std::endl;
  hipdnnDestroyConvolutionDescriptor(convDesc_);
  hipdnnDestroyFilterDescriptor(kernelDesc_);
  hipdnnDestroyTensorDescriptor(biasDesc_);
}

void ConvolutionWrapper::setConvDescriptor(int hPad,
                                           int wPad,
                                           int hStride,
                                           int wStride) {
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&convDesc_));

#if CUDNN_MAJOR > 5
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(convDesc_,
                                             hPad,
                                             wPad,
                                             hStride,
                                             wStride,
                                             1,
                                             1,  // upscales
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));
#else
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(convDesc_,
                                             hPad,
                                             wPad,
                                             hStride,
                                             wStride,
                                             1,
                                             1,  // upscales
                                             HIPDNN_CROSS_CORRELATION));
#endif
}

void ConvolutionWrapper::setKernelDescriptor(const Shape& shape) {
  ABORT_IF(shape.size() != 4,
           "CUDN requires tensors 4D. Provided {}",
           shape.toString());
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&kernelDesc_));

  int layerIn = shape[0];
  int layerOut = shape[1];
  int kernelH = shape[2];
  int kernelW = shape[3];

  CUDNN_CALL(hipdnnSetFilter4dDescriptor(kernelDesc_,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW,
                                        layerOut,
                                        layerIn,
                                        kernelH,
                                        kernelW));
}

/******************************************************************************
 * PoolingWrapper
 *****************************************************************************/

PoolingWrapper::PoolingWrapper(int height,
                               int width,
                               int padHeight,
                               int padWidth,
                               int strideHeight,
                               int strideWidth,
                               std::string mode) {
  if(mode == "max") {
    poolingMode_ = HIPDNN_POOLING_MAX;
  } else if(mode == "avg") {
    poolingMode_ = HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
  } else {
    ABORT("Unknown pooling mode.");
  }

  setPoolingDescriptor(
      height, width, padHeight, padWidth, strideHeight, strideWidth);
}

void PoolingWrapper::getOutputShape(const Shape& xShape, Shape& shape) {
  hipdnnTensorDescriptor_t xDesc;
  setCudnnTensor(xDesc, xShape);
  shape.resize(4);
  CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(poolingDesc_,
                                               xDesc,
                                               shape.data(),
                                               shape.data() + 1,
                                               shape.data() + 2,
                                               shape.data() + 3));
  hipdnnDestroyTensorDescriptor(xDesc);
}

void PoolingWrapper::forward(Tensor x, Tensor y) {
  hipSetDevice(x->getDeviceId().no);

  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, x);
  setCudnnTensor(yDesc, y);

  const float alpha = 1.0f;
  const float beta = 0.0f;

  CUDNN_CALL(hipdnnPoolingForward(cudnnHandle_,
                                 poolingDesc_,
                                 &alpha,
                                 xDesc,
                                 x->data(),
                                 &beta,
                                 yDesc,
                                 y->data()));
  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

void PoolingWrapper::backward(Tensor x, Tensor xGrad, Tensor y, Tensor yGrad) {
  hipSetDevice(x->getDeviceId().no);

  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, x);
  setCudnnTensor(yDesc, y);

  const float alpha = 1.0f;
  const float beta = 1.0f;

  CUDNN_CALL(hipdnnPoolingBackward(cudnnHandle_,
                                  poolingDesc_,
                                  &alpha,
                                  yDesc,
                                  y->data(),
                                  yDesc,
                                  yGrad->data(),
                                  xDesc,
                                  x->data(),
                                  &beta,
                                  xDesc,
                                  xGrad->data()));
  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

void PoolingWrapper::setPoolingDescriptor(int height,
                                          int width,
                                          int padHeight,
                                          int padWidth,
                                          int strideHeight,
                                          int strideWidth) {
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&poolingDesc_));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(poolingDesc_,
                                         poolingMode_,
                                         HIPDNN_NOT_PROPAGATE_NAN,
                                         height,
                                         width,
                                         padHeight,
                                         padWidth,
                                         strideHeight,
                                         strideWidth));
}

PoolingWrapper::~PoolingWrapper() {
  // std::cerr << "destroy pool-wrapper" << std::endl;
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(poolingDesc_));
}

#else

CUDNNWrapper::CUDNNWrapper() {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

CUDNNWrapper::~CUDNNWrapper() {}

ConvolutionWrapper::ConvolutionWrapper(const Shape&,
                                       const Shape&,
                                       int,
                                       int,
                                       int,
                                       int) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void ConvolutionWrapper::getOutputShape(const Shape&, Shape&) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void ConvolutionWrapper::forward(Tensor, Tensor, Tensor, Tensor) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void ConvolutionWrapper::backward(Tensor,
                                  Tensor,
                                  Tensor,
                                  Tensor,
                                  Tensor,
                                  Tensor) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

ConvolutionWrapper::~ConvolutionWrapper() {}

PoolingWrapper::PoolingWrapper(int, int, int, int, int, int, std::string) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void PoolingWrapper::getOutputShape(const Shape&, Shape&) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void PoolingWrapper::forward(Tensor x, Tensor y) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void PoolingWrapper::backward(Tensor, Tensor, Tensor, Tensor) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

PoolingWrapper::~PoolingWrapper() {}

#endif
}  // namespace marian
