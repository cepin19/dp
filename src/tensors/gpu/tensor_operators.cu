#include "hip/hip_runtime.h"
//#include <thrust/transform_reduce.h>

#include "tensors/tensor_operators.h"

#include "functional/functional.h"
#include "functional/tensor.h"
#include "tensors/gpu/backend.h"
#include "tensors/gpu/cuda_helpers.h"

#include "3rd_party/reduce_all.h"

namespace marian {

namespace gpu {

struct isnan_test {
  __host__ __device__ bool operator()(const float a) const { return isnan(a); }
};

__device__ inline float stableSigmoid(float x) {
  if(x >= 0) {
    float z = expf(-x);
    return 1.0 / (1.0 + z);
  } else {
    float z = expf(x);
    return z / (1.0 + z);
  }
}

template <typename T>
__global__ void gIsNan(T* in, int length, bool* isNan, bool* isInf, bool zero) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      if(isnan((float)in[index])) {
        if(zero) in[index] = (T)0.f;
        *isNan = true;
      }
      else if(isinf((float)in[index])) {
        if(zero) in[index] = (T)0.f;
        *isInf = true;
      }
    }
  }
}

void IsNan(Tensor in, Ptr<Allocator> allocator, bool& isNan, bool& isInf, bool zero) {
  hipSetDevice(in->getDeviceId().no);

  int length = in->size();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  auto mem = allocator->alloc<bool>(2);
  bool* dIsNan = &mem->data<bool>()[0];
  bool* dIsInf = &mem->data<bool>()[1];
  fill(in->getBackend(), dIsNan, dIsNan + 2, false);

  if(in->type() == Type::float32) {
    gIsNan<<<blocks, threads>>>(in->data<float>(), length, dIsNan, dIsInf, zero);
  } else {
    ABORT("IsNan for type {} not implemented", in->type());
  }

  CudaCopy(dIsNan, dIsNan + 1, &isNan);
  CudaCopy(dIsInf, dIsInf + 1, &isInf);

  allocator->free(mem);

  hipStreamSynchronize(0);
}

void ConcatCont(Tensor out, const std::vector<Tensor>& inputs, int axis) {
  hipSetDevice(out->getDeviceId().no);
  int step = 1;
  for(int i = 0; i < axis; ++i)
    step *= out->shape()[i];

  size_t offset1 = 0;
  for(int i = 0; i < step; ++i) {
    for(auto in : inputs) {
      size_t size = in->shape().elements() / step;
      size_t offset2 = i * size;

      hipMemcpy(out->data() + offset1,
                 in->data() + offset2,
                 size * sizeof(float),
                 hipMemcpyDeviceToDevice);

      offset1 += size;
    }
  }
  hipStreamSynchronize(0);
}

template <bool add>
__global__ void gInsertCols(float* out,
                            const float* in,
                            size_t rows,
                            size_t cols,
                            size_t cols_out,
                            size_t cols_in,
                            size_t offset_out,
                            size_t offset_in) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols_out + offset_out;
      const float* rowIn = in + j * cols_in + offset_in;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          if(add)
            rowOut[i] += rowIn[i];
          else
            rowOut[i] = rowIn[i];
      }
    }
  }
}

void Concatenate1(Tensor out, const std::vector<Tensor>& inputs) {
  hipSetDevice(out->getDeviceId().no);

  int rows = out->shape().elements() / out->shape().back();

  size_t offset = 0;
  int cols_out = out->shape().back();

  for(auto in : inputs) {
    ABORT_IF(rows != in->shape().elements() / in->shape().back(),
             "First dimension must be equal");
    int cols_in = in->shape().back();

    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols_in);

    gInsertCols<false><<<blocks, threads>>>(
        out->data(), in->data(), rows, cols_in, cols_out, cols_in, offset, 0);
    offset += cols_in;
  }
  hipStreamSynchronize(0);
}

__global__ void gJoin2(float* out,
                       size_t rowBatch,
                       size_t cols,
                       const float* in1,
                       size_t inStride1,
                       const float* in2,
                       size_t inStride2) {
  int outStride = inStride1 + inStride2;
  int rows = rowBatch * outStride;

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols;

      int curBatch = j / outStride;
      int curPos = j % outStride;

      int jIn1 = (curBatch * inStride1) + curPos;
      int jIn2 = (curBatch * inStride2) + curPos - inStride1;

      const float* rowIn1 = in1 + jIn1 * cols;
      const float* rowIn2 = in2 + jIn2 * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          if(curPos < inStride1)
            rowOut[i] = rowIn1[i];
          else
            rowOut[i] = rowIn2[i];
        }
      }
    }
  }
}

void Concatenate2(Tensor out, Tensor in1, Tensor in2) {
  hipSetDevice(out->getDeviceId().no);

  size_t rows = out->shape().elements() / out->shape().back();
  size_t cols = out->shape().back();

  size_t rowStride1 = in1->shape()[-2];
  size_t rowStride2 = in2->shape()[-2];

  size_t rowBatch = rows / out->shape()[-2];

  int blocks = std::min(MAX_BLOCKS, (int)rows);
  int threads = std::min(MAX_THREADS, (int)cols);

  gJoin2<<<blocks, threads>>>(out->data(),
                              rowBatch,
                              cols,
                              in1->data(),
                              rowStride1,
                              in2->data(),
                              rowStride2);

  hipStreamSynchronize(0);
}

void Concatenate(Tensor out, const std::vector<Tensor>& inputs, int ax) {
  if(ax == out->shape().size() - 1)
    Concatenate1(out, inputs);
  else if(ax == out->shape().size() - 2 && inputs.size() == 2)
    Concatenate2(out, inputs[0], inputs[1]);
  else
    ConcatCont(out, inputs, ax);
}

void Split1(std::vector<Tensor>& outputs, const Tensor in) {
  hipSetDevice(in->getDeviceId().no);

  size_t offset = 0;
  int rows = in->shape().elements() / in->shape().back();
  int cols_in = in->shape().back();
  for(auto out : outputs) {
    ABORT_IF(rows != out->shape().elements() / out->shape().back(),
             "First dimension must be equal");
    int cols_out = out->shape().back();

    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols_out);

    gInsertCols<true><<<blocks, threads>>>(
        out->data(), in->data(), rows, cols_out, cols_out, cols_in, 0, offset);
    offset += cols_out;
  }
  hipStreamSynchronize(0);
}

// @TODO: this function is just a temporary fix until I come up with
// something better for the situation below.
__global__ void gAddRow(float* out, const float* in, int length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      out[index] = in[index] + out[index];
    }
  }
}

void SplitCont(std::vector<Tensor>& outputs, const Tensor in, int axis) {
  hipSetDevice(in->getDeviceId().no);

  int step = 1;
  for(int i = 0; i < axis; ++i)
    step *= in->shape()[i];

  int offset1 = 0;
  for(int i = 0; i < step; ++i) {
    for(auto out : outputs) {
      int size = out->shape().elements() / step;
      int offset2 = i * size;

      // BUG: this is does not add gradients
      // hipMemcpyAsync(out->data() + offset2,
      //                in->data() + offset1,
      //                size * sizeof(float),
      //                hipMemcpyDeviceToDevice);

      // @TODO: this is a quick but bad fix for the above bug
      int threads = std::min(MAX_THREADS, size);
      int blocks = std::min(MAX_BLOCKS, size / threads + (size % threads != 0));

      gAddRow<<<blocks, threads>>>(
          out->data() + offset2, in->data() + offset1, size);
      offset1 += size;
    }
  }
  hipStreamSynchronize(0);
}

void Deconcatenate(std::vector<Tensor>& outputs, const Tensor in, int ax) {
  if(ax == in->shape().size() - 1)
    Split1(outputs, in);
  else
    SplitCont(outputs, in, ax);
}

template <bool add>
__global__ void gTransposeND(
    functional::Tensor<float> out,
    const functional::Tensor<float> in,
    const functional::Array<int, functional::Shape::size()> permute) {
  constexpr size_t N = functional::Shape::size();
  functional::Array<int, N> oDims;
  functional::Array<int, N> pDims;

  int length = out.shape().elements();
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      out.shape().dims(index, oDims);
      for(int i = 0; i < N; ++i)
        pDims[permute[i]] = oDims[i];
      if(add)
        out[index] += in[pDims];
      else
        out[index] = in[pDims];
    }
  }
}

template <bool add>
__global__ void gTranspose0213(float* out,
                               const float* in,
                               int rows,
                               int cols,
                               int stride1,
                               int stride2) {
  int stride = stride1 * stride2;
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols;

      int z = j / stride;
      int y = (j % stride) / stride1;
      int x = (j % stride) % stride1;
      int j2 = z * stride + x * stride2 + y;

      const float* rowIn = in + j2 * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          if(add)
            rowOut[i] += rowIn[i];
          else
            rowOut[i] = rowIn[i];
        }
      }
    }
  }
}

void TransposeND(Tensor out, Tensor in, const std::vector<int>& vAxis) {
  hipSetDevice(out->getDeviceId().no);
  if(vAxis == std::vector<int>({0, 2, 1, 3})) {
    int rows = out->shape().elements() / out->shape().back();
    int cols = out->shape().back();

    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols);

    int stride1 = out->shape()[-2];
    int stride2 = out->shape()[-3];

    gTranspose0213<false><<<blocks, threads>>>(
        out->data(), in->data(), rows, cols, stride1, stride2);
  } else {
    functional::Array<int, functional::Shape::size()> axes;
    int diff = functional::Shape::size() - vAxis.size();
    for(int i = 0; i < axes.size(); ++i)
      if(i < diff)
        axes[i] = i;
      else
        axes[i] = vAxis[i - diff] + diff;

    int length = out->shape().elements();
    int threads = std::min(MAX_THREADS, length);
    int blocks
        = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

    gTransposeND<false><<<blocks, threads>>>(out, in, axes);
  }
}

void TransposeNDGrad(Tensor out, Tensor in, const std::vector<int>& vAxis) {
  hipSetDevice(out->getDeviceId().no);
  if(vAxis == std::vector<int>({0, 2, 1, 3})) {
    int rows = out->shape().elements() / out->shape().back();
    int cols = out->shape().back();

    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols);

    int stride1 = out->shape()[-2];
    int stride2 = out->shape()[-3];

    gTranspose0213<true><<<blocks, threads>>>(
        out->data(), in->data(), rows, cols, stride1, stride2);
  } else {
    functional::Array<int, functional::Shape::size()> axes;
    int diff = functional::Shape::size() - vAxis.size();
    for(int i = 0; i < axes.size(); ++i)
      if(i < diff)
        axes[i] = i;
      else
        axes[i] = vAxis[i - diff] + diff;

    int length = out->shape().elements();
    int threads = std::min(MAX_THREADS, length);
    int blocks
        = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

    gTransposeND<true><<<blocks, threads>>>(out, in, axes);
  }
}

// Computes the softmax
// in - input tensor
// out - output tensor
// we compute the softmax over the the cols (last dimension)
// rows are time, batch or beam dimensions
// number of threads is number of cols or MAX_THREADS
// number of blocks is number of rows or MAX_BLOCKS
__global__ void gSoftmax(float* out,
                         functional::Shape outShape,
                         const float* in) {
  int rows = outShape.elements() / outShape.back();
  int cols = outShape.back();

  for(int bid = 0; bid < rows; bid += gridDim.x) { // loop over blocks of rows
    int j = bid + blockIdx.x; // blockIdx.x - row index (within block of rows)
    if(j < rows) { // compute softmax over one row, row elements distributed over threads
      float* so = out + j * cols; // pointer to row input data
      const float* sp = in + j * cols;

      extern __shared__ float _share[];

      // determine max (used below to improve numeric stability)
      float* _max = _share;
      _max[threadIdx.x] = -CUDA_FLT_MAX; // [threadIdx.x = relative column index within a block of columns]
      // find max over column indices that have the same relative column index (=threadIdx.x) across all blocks of columns
      for(int tid = 0; tid < cols; tid += blockDim.x) { // loop over blocks of columns, blockDim.x = index of block of columns
        // threadIdx.x = column index within block of columns; we reduce over columns within a block, then over blocks
        int i = tid + threadIdx.x;
        if(i < cols) {
          if(sp[i] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[i];
        }
      }
      __syncthreads();
      // max over columns within a column block via tree reduction
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads(); // @TODO: do we need this?

      // compute denominator
      float* _sum = _share;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          // @TODO: is it faster to cache the result of expf() in GPU RAM, or would it be faster to recompute it below?
          float ex = __expf(sp[i] - max);
          so[i] = ex;
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      // now reduce over all columns within the block
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // produce final output data
      float sum = _sum[0];
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          so[i] = so[i] / sum;
        }
      }
    }
    __syncthreads();
  }
}

void Softmax(Tensor out, Tensor in) {
  hipSetDevice(out->getDeviceId().no);

  size_t m = out->shape().elements() / out->shape().back();
  size_t k = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads;

  gSoftmax<<<blocks, threads, shared>>>(out->data(), out->shape(), in->data());
}

// @TODO: refactor to reuse code from softmax, add comments
__global__ void gLogSoftmax(float* out,
                            const functional::Shape outShape,
                            const float* in) {
  int rows = outShape.elements() / outShape.back();
  int cols = outShape.back();

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      extern __shared__ float _share[];

      float* _max = _share;
      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if(sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float sm = sp[id] - max;
          float ex = __expf(sm);
          so[id] = sm;
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols)
          so[id] -= __logf(_sum[0]);
      }
    }
    __syncthreads();
  }
}

void LogSoftmax(Tensor out, Tensor in) {
  hipSetDevice(out->getDeviceId().no);

  size_t m = out->shape().elements() / out->shape().back();
  size_t k = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads;

  gLogSoftmax<<<blocks, threads, shared>>>(
      out->data(), out->shape(), in->data());
}

///////////////////////////////////////////////////////

__global__ void gSoftmaxGrad(float* grad,
                             const float* adj,
                             const float* val,
                             const int rows,
                             const int cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share;

      float* gradRow = grad + j * cols;
      const float* adjRow = adj + j * cols;
      const float* valRow = val + j * cols;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += valRow[id] * adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float val = valRow[id] * (adjRow[id] - _sum[0]);
          if(val)
            gradRow[id] += val;
        }
      }
    }
    __syncthreads();
  }
}

// @TODO: refactor with logsoftmax, add math
void SoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
  hipSetDevice(adj->getDeviceId().no);
  // grad and val are both m-by-k matrices, passed as input.
  // A weighted average of each row of grad (according to the weights
  // specified in val) is computed and subtracted from Out.
  // adj is multiplied for each element to get backward step in autodiff
  int m = grad->shape().elements() / grad->shape().back();
  int k = grad->shape().back();

  int blocks = std::min(MAX_BLOCKS, m);
  int threads = std::min(MAX_THREADS, k);
  int shared = sizeof(float) * threads;
  gSoftmaxGrad<<<blocks, threads, shared>>>(
      grad->data(), adj->data(), val->data(), m, k);
}

__global__ void gLogSoftmaxGrad(float* grad,
                                const float* adj,
                                const float* val,
                                const int rows,
                                const int cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share;

      float* gradRow = grad + j * cols;
      const float* adjRow = adj + j * cols;
      const float* valRow = val + j * cols;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols)
          gradRow[id] += adjRow[id] - (expf(valRow[id]) * _sum[0]);
      }
    }
    __syncthreads();
  }
}

void LogSoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
  hipSetDevice(adj->getDeviceId().no);

  // grad and val are both m-by-k matrices, passed as input.
  // A weighted average of each row of grad (according to the weights
  // specified in val) is computed and subtracted from Out.
  // adj is multiplied for each element to get backward step in autodiff
  int m = grad->shape().elements() / grad->shape().back();
  int k = grad->shape().back();

  int blocks = std::min(MAX_BLOCKS, m);
  int threads = std::min(MAX_THREADS, k);
  int shared = sizeof(float) * threads;
  gLogSoftmaxGrad<<<blocks, threads, shared>>>(
      grad->data(), adj->data(), val->data(), m, k);
}

///////////////////////////////////////////////////////
__global__ void gArgmax(float* out,
                        const float* data,
                        size_t rows,
                        size_t cols) {
  size_t row = blockIdx.x;
  size_t startInd = row * cols;
  float maxScore = -99999;
  size_t maxInd;
  for(size_t col = 0; col < cols; ++col) {
    size_t ind = startInd + col;
    float score = data[ind];
    if(score > maxScore) {
      maxScore = score;
      maxInd = col;
    }
  }
  out[row] = maxInd;
}

///////////////////////////////////////////////////////

__global__ void gCopyRows(float* out,
                          const float* in,
                          size_t cols,
                          const IndexType* sourceRowIdx,
                          size_t rows) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      size_t dstId = j;
      size_t srcId = sourceRowIdx[j];

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

void CopyRows(Tensor out,
              const Tensor in,
              const Tensor indices) {

  matchOrAbort<IndexType>(indices->type());

  hipSetDevice(out->getDeviceId().no);

  size_t cols = in->shape().back();
  size_t rowsToCopy = indices->size();

  int threads = std::min(MAX_THREADS, (int)cols);
  int blocks = std::min(MAX_BLOCKS, (int)rowsToCopy);

  gCopyRows<<<blocks, threads>>>(
      out->data(), in->data(), cols, indices->data<IndexType>(), rowsToCopy);
}

__global__ void gPasteRows(float* out,
                           const float* in,
                           size_t cols,
                           const IndexType* targetRowIdx,
                           size_t rows) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      size_t dstId = targetRowIdx[j];
      size_t srcId = j;

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          atomicAdd(rowOut + i, rowIn[i]);
      }
    }
  }
}

void PasteRows(Tensor out,
               const Tensor in,
               const Tensor indices) {

  matchOrAbort<IndexType>(indices->type());

  hipSetDevice(out->getDeviceId().no);

  size_t cols = in->shape().back();
  size_t rowsToCopy = indices->size();

  int threads = std::min(MAX_THREADS, (int)cols);
  int blocks = std::min(MAX_BLOCKS, (int)rowsToCopy);

  gPasteRows<<<blocks, threads>>>(
      out->data(), in->data(), cols, indices->data<IndexType>(), rowsToCopy);
}

/////////////

__global__ void gCopyCols(float* out,
                          const float* in,
                          size_t rows,
                          size_t colsIn,
                          const IndexType* sourceColIdx,
                          size_t colsOut) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* rowIn = in + j * colsIn;
      float* rowOut = out + j * colsOut;

      for(int tid = 0; tid < colsOut; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < colsOut)
          rowOut[i] = rowIn[sourceColIdx[i]];
      }
    }
  }
}

void CopyCols(Tensor out, const Tensor in, const Tensor indices) {
  matchOrAbort<IndexType>(indices->type());

  hipSetDevice(out->getDeviceId().no);

  size_t rows = in->shape().elements() / in->shape().back();
  size_t cols = in->shape().back();

  size_t colsToCopy = indices->size();

  int threads = std::min(MAX_THREADS, (int)colsToCopy);
  int blocks = std::min(MAX_BLOCKS, (int)rows);

  gCopyCols<<<blocks, threads>>>(
      out->data(), in->data(), rows, cols, indices->data<IndexType>(), colsToCopy);
}

__global__ void gPasteCols(float* out,
                           const float* in,
                           size_t rows,
                           size_t colsOut,
                           const IndexType* targetColIdx,
                           size_t colsIn) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* rowIn = in + j * colsIn;
      float* rowOut = out + j * colsOut;

      for(int tid = 0; tid < colsIn; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < colsIn)
          rowOut[targetColIdx[i]] += rowIn[i];
      }
    }
  }
}

void PasteCols(Tensor out,
               const Tensor in,
               const Tensor indices) {
  matchOrAbort<IndexType>(indices->type());

  hipSetDevice(out->getDeviceId().no);

  size_t rows = in->shape().elements() / in->shape().back();
  size_t cols = in->shape().back();

  size_t colsToCopy = indices->size();

  int threads = std::min(MAX_THREADS, (int)colsToCopy);
  int blocks = std::min(MAX_BLOCKS, (int)rows);

  gPasteCols<<<blocks, threads>>>(
      out->data(), in->data(), rows, cols, indices->data<IndexType>(), colsToCopy);
}

__global__ void gSelect(float* out,
                        functional::Shape outShape,
                        const float* in,
                        const functional::Shape inShape,
                        int axis,
                        IndexType* d_indices) {
  int length = outShape.elements();
  functional::Array<int, functional::Shape::size()> dims;

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      outShape.dims(index, dims);
      dims[axis] = d_indices[dims[axis]];
      int inIndex = inShape.index(dims);
      out[index] = in[inIndex];
    }
  }
}

__global__ void gInsert(float* out,
                        functional::Shape outShape,
                        const float* in,
                        const functional::Shape inShape,
                        int axis,
                        IndexType* d_indices) {
  int length = inShape.elements();
  functional::Array<int, functional::Shape::size()> dims;

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      inShape.dims(index, dims);
      dims[axis] = d_indices[dims[axis]];
      int outIndex = outShape.index(dims);
      out[outIndex] += in[index];
    }
  }
}

void Select(Tensor out,
            const Tensor in,
            const Tensor indices,
            int axis) {
  matchOrAbort<IndexType>(indices->type());

  hipSetDevice(out->getDeviceId().no);

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  int axisGPU = axis + functional::Shape::size() - out->shape().size();
  gSelect<<<blocks, threads>>>(out->data(),
                               out->shape(),
                               in->data(),
                               in->shape(),
                               axisGPU,
                               indices->data<IndexType>());
}

void Insert(Tensor out,
            const Tensor in,
            const Tensor indices,
            int axis) {
  matchOrAbort<IndexType>(indices->type());
  hipSetDevice(in->getDeviceId().no);

  int length = in->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  int axisGPU = axis + functional::Shape::size() - out->shape().size();
  gInsert<<<blocks, threads>>>(out->data(),
                               out->shape(),
                               in->data(),
                               in->shape(),
                               axisGPU,
                               indices->data<IndexType>());
}

__global__ void gGRUFastForward(float* out,
                                const float* state,
                                const float* xW,
                                const float* sU,
                                const float* b,
                                const float* mask,
                                size_t rows,
                                size_t cols,
                                bool final) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];
      float* rowOut = out + j * cols;
      const float* rowState = state + j * cols;

      const float* xWrow = xW + j * cols * 3;
      const float* sUrow = sU + j * cols * 3;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          float r = stableSigmoid(xWrow[i] + sUrow[i] + b[i]);

          int k = i + cols;

          float z = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

          int l = i + 2 * cols;
          float h;
          if(final)
            h = tanhf(xWrow[l] + (sUrow[l] + b[l]) * r);
          else
            h = tanhf(xWrow[l] + sUrow[l] * r + b[l]);

          float out = (1.0f - z) * h + z * rowState[i];
          rowOut[i] = m * out + (1 - m) * rowState[i];
        }
      }
    }
  }
}

void GRUFastForward(Tensor out, std::vector<Tensor> inputs, bool final) {
  hipSetDevice(out->getDeviceId().no);

  int rows = out->shape().elements() / out->shape().back();
  int cols = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gGRUFastForward<<<blocks, threads>>>(
      out->data(),                                // output
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      rows,
      cols,
      final);
}

__global__ void gGRUFastBackward(float* outState,
                                 float* outXW,
                                 float* outSU,
                                 float* outB,
                                 const float* state,
                                 const float* xW,
                                 const float* sU,
                                 const float* b,
                                 const float* mask,
                                 const float* adj,
                                 size_t rows,
                                 size_t cols,
                                 bool final) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOutState = outState + j * cols;
      float* rowOutXW = outXW + j * cols * 3;
      float* rowOutSU = outSU + j * cols * 3;

      const float* rowState = state + j * cols;
      const float* rowXW = xW + j * cols * 3;
      const float* rowSU = sU + j * cols * 3;
      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          int k = i + cols;
          int l = i + 2 * cols;

          float r = stableSigmoid(rowXW[i] + rowSU[i] + b[i]);
          float z = stableSigmoid(rowXW[k] + rowSU[k] + b[k]);

          float h;
          if(final)
            h = tanhf(rowXW[l] + (rowSU[l] + b[l]) * r);
          else
            h = tanhf(rowXW[l] + rowSU[l] * r + b[l]);

          float adj = rowAdj[i];

          float t = (1 - z) * (1 - h * h);

          // df/ds
          if(outState)
            rowOutState[i] += (m * z - m + 1) * adj;

          // df/d(xW_r) ...
          float dfdxW_r = m * r * (1 - r) * t * adj;
          if(final)
            dfdxW_r *= rowSU[l] + b[l];
          else
            dfdxW_r *= rowSU[l];
          if(outXW)
            rowOutXW[i] += dfdxW_r;
          if(outSU)
            rowOutSU[i] += dfdxW_r;
          if(outB)
            atomicAdd(outB + i, dfdxW_r);

          // df/d(xW_z) ...
          float dfdxW_z = m * (1 - z) * z * (rowState[i] - h) * adj;
          if(outXW)
            rowOutXW[k] += dfdxW_z;
          if(outSU)
            rowOutSU[k] += dfdxW_z;
          if(outB)
            atomicAdd(outB + k, dfdxW_z);

          // df/d(xW_x) ...
          float dfdxW_x = m * t * adj;
          if(outXW)
            rowOutXW[l] += dfdxW_x;
          if(outSU)
            rowOutSU[l] += dfdxW_x * r;
          if(outB)
            if(final)
              atomicAdd(outB + l, dfdxW_x * r);
            else
              atomicAdd(outB + l, dfdxW_x);
        }
      }
    }
  }
}

void GRUFastBackward(std::vector<Tensor> outputs,
                     std::vector<Tensor> inputs,
                     Tensor adj,
                     bool final) {
  hipSetDevice(adj->getDeviceId().no);

  int rows = adj->shape().elements() / adj->shape().back();
  int cols = adj->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gGRUFastBackward<<<blocks, threads>>>(
      outputs[0] ? outputs[0]->data() : 0,        // state - adj
      outputs[1] ? outputs[1]->data() : 0,        // xW - adj
      outputs[2] ? outputs[2]->data() : 0,        // sU - adj
      outputs[3] ? outputs[3]->data() : 0,        // b - adj
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      adj->data(),
      rows,
      cols,
      final);
}

__global__ void gCrossEntropyPick(float* out,
                                  const functional::Shape outShape,
                                  const float* in,
                                  const functional::Shape inShape,
                                  const IndexType* pick) {
  int rows = inShape.elements() / inShape.back();
  int cols = inShape.back();

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* sp = in + j * cols;

      extern __shared__ float _share[];
      float* _max = _share;

      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if(sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += __expf(sp[id] - max);
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // cross-entropy
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id == (int)pick[j]) {
          out[j] = __logf(_sum[0]) - sp[id] + max;
        }
      }
    }
    __syncthreads();
  }
}

// In each j-th row, take the corresponding j-th label index i from indices and compute:
// For each vocabulary item v, the only non-zero element in a row in the sum is the item
// that matches the label indexed by i (the picked element).
// C = sum_{v in V}(-logsoftmax(A) * delta(v, i) = -logsoftmax(A)[i]
void CrossEntropyPick(Tensor out, Tensor in, Tensor indices) {
  matchOrAbort<IndexType>(indices->type());

  hipSetDevice(out->getDeviceId().no);

  int rows = in->shape().elements() / in->shape().back();
  int cols = in->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)rows);
  int threads = std::min(MAX_THREADS, (int)cols);
  int shared = sizeof(float) * threads;

  gCrossEntropyPick<<<blocks, threads, shared>>>(
      out->data(), out->shape(), in->data(), in->shape(), indices->data<IndexType>());
}

__global__ void gCrossEntropyPickBackward(float* out,
                                          const functional::Shape outShape,
                                          const float* adj,
                                          const float* in,
                                          const IndexType* pick) {
  int rows = outShape.elements() / outShape.back();
  int cols = outShape.back();
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* sp = in + j * cols;
      float* so = out + j * cols;

      extern __shared__ float _share[];
      float* _max = _share;

      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if(sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = __expf(sp[id] - max);
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // cross-entropy
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float sub = (float)(id == (int)pick[j]);
          so[id] += adj[j] * (__expf(sp[id] - max) / _sum[0] - sub);
        }
      }
    }
    __syncthreads();
  }
}

void CrossEntropyPickBackward(Tensor out, Tensor adj, Tensor a, Tensor indices) {
  matchOrAbort<IndexType>(indices->type());

  hipSetDevice(out->getDeviceId().no);

  int rows = out->shape().elements() / out->shape().back();
  int cols = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)rows);
  int threads = std::min(MAX_THREADS, (int)cols);
  int shared = sizeof(float) * threads;

  gCrossEntropyPickBackward<<<blocks, threads, shared>>>(
      out->data(), out->shape(), adj->data(), a->data(), indices->data<IndexType>());
}

float L2Norm(Tensor in) {
  hipSetDevice(in->getDeviceId().no);

  int size = in->shape().elements();
  int threads = std::min(MAX_THREADS, size);
  int blocks = std::min(MAX_BLOCKS, size / threads + (size % threads != 0));

  uint8_t* data;
  hipMalloc(&data, blocks * sizeof(float));
  Tensor out(new TensorBase(New<MemoryPiece>(data, blocks * sizeof(float)),
                            {1, blocks},
                            in->getBackend()));

  using namespace functional;
  ReduceAll(_1 * _1, out, in);
  float dataCpu = sqrtf(out->get(0));
  out.reset();
  hipFree(data);
  return dataCpu;
}

__global__ void gAtt(float* out,
                     const float* va,
                     const float* ctx,
                     const float* state,
                     int m,  // total rows (batch x time x beam)
                     int k,  // depth
                     int b,  // batch size
                     int t   // time of ctx
) {
  int rows = m;
  int cols = k;

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* vaRow = va;
      const float* ctxRow = ctx + (j % (b * t)) * cols;
      const float* stateRow = state + ((j / (b * t)) * b + j % b) * cols;

      extern __shared__ float _share[];
      float* _sum = _share;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float z = ctxRow[id] + stateRow[id];
          float ex = tanhf(z) * vaRow[id];
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      out[j] = _sum[0];
    }
    __syncthreads();
  }
}

void Att(Tensor out, Tensor va, Tensor context, Tensor state) {
  hipSetDevice(out->getDeviceId().no);

  size_t m = out->shape().elements() / out->shape().back();
  size_t k = context->shape()[-1];
  size_t b = context->shape()[-2];
  size_t t = context->shape()[-3];

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads;

  gAtt<<<blocks, threads, shared>>>(
      out->data(), va->data(), context->data(), state->data(), m, k, b, t);
}

__global__ void gAttBack(float* gVa,
                         float* gContext,
                         float* gState,
                         const float* va,
                         const float* context,
                         const float* state,
                         const float* adj,
                         int m,  // rows
                         int k,  // cols
                         int n   // batch size
) {
  int rows = m;
  int cols = k;
  for(int bid = 0; bid < m; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* gcRow = gContext + j * cols;
      float* gsRow = gState + (j % n) * cols;

      const float* cRow = context + j * cols;
      const float* sRow = state + (j % n) * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float z = cRow[id] + sRow[id];

          float t = tanhf(z);
          float r = va[id] * (1.f - t * t);

          gcRow[id] += r * adj[j];
          gsRow[id] += r * adj[j];
          atomicAdd(gVa + id, t * adj[j]);
        }
      }
    }
  }
}

void AttBack(Tensor gVa,
             Tensor gContext,
             Tensor gState,
             Tensor va,
             Tensor context,
             Tensor state,
             Tensor adj) {
  hipSetDevice(adj->getDeviceId().no);

  size_t m = adj->shape().elements() / adj->shape()[-1];
  size_t k = context->shape()[-1];
  size_t n = context->shape()[-2];

  int blocks = std::min(MAX_BLOCKS, (int)n);
  int threads = std::min(MAX_THREADS, (int)k);

  gAttBack<<<blocks, threads>>>(gVa->data(),
                                gContext->data(),
                                gState->data(),

                                va->data(),
                                context->data(),
                                state->data(),

                                adj->data(),
                                m,
                                k,
                                n);
}

__global__ void gLNormalization(float* out,
                                const float* in,
                                const float* alpha,
                                const float* beta,
                                int rows,
                                int cols,
                                float eps = 1e-9) {
  extern __shared__ float _share[];

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      float* _sum = _share;
      _sum[threadIdx.x] = 0.0f;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float mean = _sum[0] / cols;
      __syncthreads();

      float* _sqSum = _share;

      _sqSum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = sp[id] - mean;
          _sqSum[threadIdx.x] += ex * ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sqSum[threadIdx.x] += _sqSum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float sigma = sqrtf(eps + (_sqSum[0] / cols));
      __syncthreads();

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float t = alpha[id] * ((sp[id] - mean) / sigma);
          if(beta != nullptr)
            t += beta[id];
          so[id] = t;
        }
      }
    }
    __syncthreads();
  }
}

void LayerNormalization(Tensor out,
                        Tensor in,
                        Tensor gamma,
                        Tensor beta,
                        float eps) {
  hipSetDevice(out->getDeviceId().no);

  int rows = in->shape().elements() / in->shape().back();
  int cols = in->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)rows);
  int threads = std::min(MAX_THREADS, (int)cols);
  int shared = threads * sizeof(float);

  gLNormalization<<<blocks, threads, shared>>>(out->data(),
                                               in->data(),
                                               gamma->data(),
                                               beta ? beta->data() : nullptr,
                                               rows,
                                               cols,
                                               eps);
}

__global__ void gLayerNormalizationGrad(float* gradX,
                                        float* gradGamma,
                                        float* gradBeta,
                                        float* adj,
                                        float* y,
                                        float* x,
                                        float* gamma,
                                        float* beta,
                                        int rows,
                                        int cols,
                                        float eps = 1e-9) {
  extern __shared__ float shared[];

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* sum_adj = shared;
      float* sum_adj_x = shared + blockDim.x;
      float* sum_x = shared + 2 * blockDim.x;
      float* sum_sqr = shared + 3 * blockDim.x;

      const float* xRow = x + j * cols;
      const float* yRow = y + j * cols;
      const float* adjRow = adj + j * cols;
      float* gradXRow = gradX + j * cols;

      sum_x[threadIdx.x] = 0.0f;
      sum_adj[threadIdx.x] = 0.0f;
      sum_adj_x[threadIdx.x] = 0.0f;
      sum_sqr[threadIdx.x] = 0.0f;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          sum_x[threadIdx.x] += xRow[id];
          sum_adj_x[threadIdx.x]
              += adjRow[id] * (yRow[id] - ((beta) ? beta[id] : 0)) / gamma[id];
          sum_adj[threadIdx.x] += adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          sum_x[threadIdx.x] += sum_x[threadIdx.x + skip];
          sum_adj[threadIdx.x] += sum_adj[threadIdx.x + skip];
          sum_adj_x[threadIdx.x] += sum_adj_x[threadIdx.x + skip];
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float mean = sum_x[0] / cols;
      __syncthreads();

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = xRow[id] - mean;
          sum_sqr[threadIdx.x] += ex * ex;
        }
      }

      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          sum_sqr[threadIdx.x] += sum_sqr[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float sigma = sqrtf(eps + (sum_sqr[0] / cols));
      __syncthreads();

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float grad_x = 0.0f;
          float x_hat = (yRow[id] - ((beta) ? beta[id] : 0)) / gamma[id];
          grad_x += cols * adjRow[id];
          grad_x -= sum_adj[0];
          grad_x -= sum_adj_x[0] * x_hat;
          grad_x /= (cols * sigma);

          float valX = gamma[id] * grad_x;
          float sign = (0.f < valX) - (valX < 0.f);
          valX = fabs(valX) > 1000 ? sign * 1000 : valX;

          gradXRow[id] += valX;
          atomicAdd(gradGamma + id, adjRow[id] * x_hat);
          if(beta) {
            atomicAdd(gradBeta + id, adjRow[id]);
          }
        }
      }
    }
    __syncthreads();
  }
}

void LayerNormalizationGrad(Tensor gradX,
                            Tensor gradGamma,
                            Tensor gradBeta,
                            Tensor adj,
                            Tensor y,
                            Tensor x,
                            Tensor gamma,
                            Tensor beta,
                            float eps) {
  hipSetDevice(adj->getDeviceId().no);
  int rows = y->shape().elements() / y->shape()[-1];
  int cols = y->shape()[-1];

  int threads = std::min(MAX_THREADS, cols);
  int blocks = std::min(MAX_BLOCKS, rows);
  int shared = sizeof(float) * threads * 4;

  gLayerNormalizationGrad<<<blocks, threads, shared>>>(
      gradX->data(),
      gradGamma->data(),
      (gradBeta) ? gradBeta->data() : nullptr,
      adj->data(),
      y->data(),
      x->data(),
      gamma->data(),
      (beta) ? beta->data() : nullptr,
      rows,
      cols,
      eps);
}

template <bool add>
__global__ void gShift(float* out,
                       const float* in,
                       int length,
                       int offset,
                       float padValue) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      if(add) {
        if(index - offset >= 0 && index - offset < length)
          out[index] += in[index - offset];
      } else {
        if(index - offset < 0 || index - offset >= length)
          out[index] = padValue;
        else
          out[index] = in[index - offset];
      }
    }
  }
}

void Shift(Tensor out,
           Tensor in,
           marian::Shape shift,
           float padValue,
           bool invert) {
  ABORT_IF(in->shape().size() != shift.size(), "bad dimensions");

  // BUGBUG: This can only shift along the first axis. Shifting, e.g., along the
  // last axis cannot be implemented this way.
  int offset = 0;
  for(int i = 0; i < shift.size(); ++i)
    offset += in->shape().stride(i) * shift[i];

  if(invert)
    offset = -offset;

  hipSetDevice(out->getDeviceId().no);

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gShift<false>
      <<<blocks, threads>>>(out->data(), in->data(), length, offset, padValue);
}

void ShiftGrad(Tensor out, Tensor in, marian::Shape shift, bool invert) {
  ABORT_IF(in->shape().size() != shift.size(), "bad dimensions");

  // BUGBUG: This can only shift along the first axis. Shifting, e.g., along the
  // last axis cannot be implemented this way.
  int offset = 0;
  for(int i = 0; i < shift.size(); ++i)
    offset += in->shape().stride(i) * shift[i];

  if(invert)
    offset = -offset;

  hipSetDevice(out->getDeviceId().no);

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gShift<true>
      <<<blocks, threads>>>(out->data(), in->data(), length, offset, 0.f);
}

__global__ void gSetSparse(float* out,
                           const size_t* indices,
                           const float* values,
                           int length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      out[indices[index]] = values[index];
    }
  }
}

void SetSparse(float* out,
               const std::vector<size_t>& indices,
               const std::vector<float>& values) {
  int length = indices.size();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, length * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        length * sizeof(size_t),
                        hipMemcpyHostToDevice));

  float* d_values;
  CUDA_CHECK(hipMalloc(&d_values, length * sizeof(float)));
  CUDA_CHECK(hipMemcpy(
      d_values, values.data(), length * sizeof(float), hipMemcpyHostToDevice));

  gSetSparse<<<blocks, threads>>>(out, d_indices, d_values, length);

  hipFree(d_indices);
  hipFree(d_values);
}

/******************************************************************************/

__global__ void gLSTMCellForward(float* out,
                                 const float* cell,
                                 const float* xW,
                                 const float* sU,
                                 const float* b,
                                 const float* mask,
                                 size_t rows,
                                 size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOut = out + j * cols;
      const float* rowCell = cell + j * cols;

      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          float gf = stableSigmoid(xWrow[i] + sUrow[i] + b[i]);

          int k = i + cols;
          float gi = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

          int l = i + 2 * cols;
          float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);

          float cout = gf * rowCell[i] + gi * gc;
          rowOut[i] = m * cout + (1 - m) * rowCell[i];
        }
      }
    }
  }
}

void LSTMCellForward(Tensor out, std::vector<Tensor> inputs) {
  hipSetDevice(out->getDeviceId().no);

  int rows = out->shape().elements() / out->shape().back();
  int cols = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMCellForward<<<blocks, threads>>>(
      out->data(),                                // output
      inputs[0]->data(),                          // cell state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      rows,
      cols);
}

__global__ void gLSTMOutputForward(float* out,
                                   const float* cell,
                                   const float* xW,
                                   const float* sU,
                                   const float* b,
                                   size_t rows,
                                   size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols;
      const float* rowCell = cell + j * cols;

      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          int k = i + 3 * cols;
          float go = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

          rowOut[i] = go * tanhf(rowCell[i]);
        }
      }
    }
  }
}

void LSTMOutputForward(Tensor out, std::vector<Tensor> inputs) {
  hipSetDevice(out->getDeviceId().no);

  int rows = out->shape().elements() / out->shape().back();
  int cols = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMOutputForward<<<blocks, threads>>>(out->data(),        // output
                                          inputs[0]->data(),  // cell state
                                          inputs[1]->data(),  // xW
                                          inputs[2]->data(),  // sU
                                          inputs[3]->data(),  // b
                                          rows,
                                          cols);
}

__global__ void gLSTMCellBackward(float* outCell,
                                  float* outXW,
                                  float* outSU,
                                  float* outB,
                                  const float* cell,
                                  const float* xW,
                                  const float* sU,
                                  const float* b,
                                  const float* mask,
                                  const float* adj,
                                  size_t rows,
                                  size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOutCell = outCell + j * cols;
      float* rowOutXW = outXW + j * cols * 4;
      float* rowOutSU = outSU + j * cols * 4;

      const float* rowCell = cell + j * cols;
      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          float gf = stableSigmoid(xWrow[i] + sUrow[i] + b[i]);

          int k = i + cols;
          float gi = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

          int l = i + 2 * cols;
          float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);

          float adj = rowAdj[i];

          // dc/dc_{t-1}
          if(outCell)
            rowOutCell[i] += (m * gf - m + 1) * adj;

          // dc/d(b_f) = dc/d(xW_f) ...
          float dcdxf = m * rowCell[i] * gf * (1 - gf) * adj;
          if(outXW)
            rowOutXW[i] += dcdxf;
          if(outSU)
            rowOutSU[i] += dcdxf;
          if(outB)
            atomicAdd(outB + i, dcdxf);

          // dc/d(b_i) ...
          float dcdb_i = m * gc * gi * (1 - gi) * adj;
          if(outXW)
            rowOutXW[k] += dcdb_i;
          if(outSU)
            rowOutSU[k] += dcdb_i;
          if(outB)
            atomicAdd(outB + k, dcdb_i);

          // dc/d(b_c) ...
          float dcdxc = m * gi * (1 - gc * gc) * adj;
          if(outXW)
            rowOutXW[l] += dcdxc;
          if(outSU)
            rowOutSU[l] += dcdxc;
          if(outB)
            atomicAdd(outB + l, dcdxc);
        }
      }
    }
  }
}

void LSTMCellBackward(std::vector<Tensor> outputs,
                      std::vector<Tensor> inputs,
                      Tensor adj) {
  hipSetDevice(adj->getDeviceId().no);

  int rows = adj->shape().elements() / adj->shape().back();
  int cols = adj->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMCellBackward<<<blocks, threads>>>(
      outputs[0] ? outputs[0]->data() : 0,        // state - adj
      outputs[1] ? outputs[1]->data() : 0,        // xW - adj
      outputs[2] ? outputs[2]->data() : 0,        // sU - adj
      outputs[3] ? outputs[3]->data() : 0,        // b - adj
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      adj->data(),
      rows,
      cols);
}

__global__ void gLSTMOutputBackward(float* outCell,
                                    float* outXW,
                                    float* outSU,
                                    float* outB,
                                    const float* cell,
                                    const float* xW,
                                    const float* sU,
                                    const float* b,
                                    const float* adj,
                                    size_t rows,
                                    size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOutCell = outCell + j * cols;
      float* rowOutXW = outXW + j * cols * 4;
      float* rowOutSU = outSU + j * cols * 4;

      const float* rowCell = cell + j * cols;
      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          int k = i + 3 * cols;
          float go = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

          float t = tanhf(rowCell[i]);

          float adj = rowAdj[i];

          // dc/dc_{t-1}
          if(outCell)
            rowOutCell[i] += go * (1 - t * t) * adj;

          // dc/d(b_o) = dc/d(xW_f) ...
          float dcdxo = t * go * (1 - go) * adj;
          if(outXW)
            rowOutXW[k] += dcdxo;
          if(outSU)
            rowOutSU[k] += dcdxo;
          if(outB)
            atomicAdd(outB + k, dcdxo);
        }
      }
    }
  }
}

void LSTMOutputBackward(std::vector<Tensor> outputs,
                        std::vector<Tensor> inputs,
                        Tensor adj) {
  hipSetDevice(adj->getDeviceId().no);

  int rows = adj->shape().elements() / adj->shape().back();
  int cols = adj->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMOutputBackward<<<blocks, threads>>>(
      outputs[0] ? outputs[0]->data() : 0,  // state - adj
      outputs[1] ? outputs[1]->data() : 0,  // xW - adj
      outputs[2] ? outputs[2]->data() : 0,  // sU - adj
      outputs[3] ? outputs[3]->data() : 0,  // b - adj
      inputs[0]->data(),                    // state
      inputs[1]->data(),                    // xW
      inputs[2]->data(),                    // sU
      inputs[3]->data(),                    // b
      adj->data(),
      rows,
      cols);
}

__global__ void gHighwayForward(float* out,
                                const float* in1,
                                const float* in2,
                                const float* t,
                                size_t length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      float sigma = stableSigmoid(t[index]);
      out[index] = in1[index] * sigma + in2[index] * (1.f - sigma);
    }
  }
}

void HighwayForward(Tensor out,
                    const Tensor in1,
                    const Tensor in2,
                    const Tensor t) {
  hipSetDevice(out->getDeviceId().no);

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gHighwayForward<<<blocks, threads>>>(
      out->data(), in1->data(), in2->data(), t->data(), length);
}

__global__ void gHighwayBackward(float* out1,
                                 float* out2,
                                 float* outt,
                                 const float* in1,
                                 const float* in2,
                                 const float* t,
                                 const float* adj,
                                 size_t length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      float sigma = stableSigmoid(t[index]);
      out1[index] = sigma * adj[index];
      out2[index] = (1.f - sigma) * adj[index];
      outt[index]
          = sigma * (1.f - sigma) * (in1[index] - in2[index]) * adj[index];
    }
  }
}

void HighwayBackward(Tensor out1,
                     Tensor out2,
                     Tensor outt,
                     const Tensor in1,
                     const Tensor in2,
                     const Tensor t,
                     const Tensor adj) {
  hipSetDevice(out1->getDeviceId().no);

  int length = out1->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gHighwayBackward<<<blocks, threads>>>(out1->data(),
                                        out2->data(),
                                        outt->data(),
                                        in1->data(),
                                        in2->data(),
                                        t->data(),
                                        adj->data(),
                                        length);
}

__global__ void gMaxPoolingForward(float* out,
                                   int outRows,
                                   int outCols,
                                   float* in,
                                   int inRows,
                                   int inCols,
                                   float* mask,
                                   int numKernels,
                                   int maskCols,
                                   int width,
                                   int lastWidth) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if(tid >= outRows * outCols)
    return;

  int rowId = tid / outRows;
  int colId = tid % outRows;

  float* b = in + (rowId * inCols) + (colId * width);
  float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

  if(colId == outRows - 1) {
    width = lastWidth;
  }

  float currentMax = b[0] * localMask[0];
  for(int i = 1; i < width; ++i) {
    if(b[i] * localMask[i] > currentMax) {
      currentMax = b[i] * localMask[i];
    }
  }

  out[rowId + (colId * outCols)] = currentMax;
}

void PoolingWithMaskingForward(Tensor out,
                               Tensor in,
                               Tensor mask,
                               int width,
                               bool isEven) {
  int n = out->shape().elements();
  int threads = std::min(n, MAX_THREADS);
  int blocks = n / threads + (n % threads != 0);

  auto& inShape = in->shape();
  int inRows = inShape[0] * inShape[1];
  int inCols = inShape[2];

  auto& outShape = out->shape();
  int outRows = outShape[2];
  int outCols = outShape[0] * outShape[1];

  int lastWidth
      = ((inCols - isEven) % width == 0) ? width : (inCols - isEven) % width;

  gMaxPoolingForward<<<blocks, threads>>>(out->data(),
                                          outRows,
                                          outCols,
                                          in->data(),
                                          inRows,
                                          inCols,
                                          mask->data(),
                                          outShape[1],
                                          mask->shape()[2],
                                          width,
                                          lastWidth);
}

__global__ void gMaxPoolingBackward(float* adj,
                                    int adjRows,
                                    int adjCols,
                                    float* in,
                                    float* adjIn,
                                    int inRows,
                                    int inCols,
                                    float* mask,
                                    int numKernels,
                                    int maskCols,
                                    int width,
                                    int lastWidth) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if(tid >= adjRows * adjCols)
    return;

  int rowId = tid / adjRows;
  int colId = tid % adjRows;

  float* b = in + (rowId * inCols) + (colId * width);

  if(colId == adjRows - 1) {
    width = lastWidth;
  }

  float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;
  size_t currentMaxIdx = 0;
  for(int i = 1; i < width; ++i) {
    if(b[i] * localMask[i] > b[currentMaxIdx] * localMask[currentMaxIdx]) {
      currentMaxIdx = i;
    }
  }

  adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx]
      += adj[rowId + (colId * adjCols)];
}

void PoolingWithMaskingBackward(Tensor adj,
                                Tensor adjIn,
                                Tensor in,
                                Tensor mask,
                                int width,
                                bool isEven) {
  int n = adj->shape().elements();
  int threads = std::min(n, 512);
  int blocks = n / threads + (n % threads != 0);

  auto& inShape = in->shape();
  int inRows = inShape[0] * inShape[1];
  int inCols = inShape[2];

  auto& adjShape = adj->shape();
  int adjRows = adjShape[2];
  int adjCols = adjShape[0] * adjShape[1];

  int lastWidth
      = ((inCols - isEven) % width == 0) ? width : (inCols - isEven) % width;

  gMaxPoolingBackward<<<blocks, threads>>>(adj->data(),
                                           adjRows,
                                           adjCols,
                                           in->data(),
                                           adjIn->data(),
                                           inRows,
                                           inCols,
                                           mask->data(),
                                           adjShape[1],
                                           mask->shape()[2],
                                           width,
                                           lastWidth);
}
}  // namespace gpu
}  // namespace marian
