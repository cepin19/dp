#include "hip/hip_runtime.h"
/* All or part of this file was contributed by Intel under license:
 *   Copyright (C) 2017-2018 Intel Corporation
 *   SPDX-License-Identifier: MIT
 */

#include <hip/hip_runtime.h>
#include <limits>

#include "data/types.h"
#include "tensors/tensor.h"
#include "translator/helpers.h"

namespace marian {

namespace gpu {

__global__ void gSetColumn(float* d_in,
                           size_t n_columns,
                           size_t n_rows,
                           size_t noColumn,
                           float value) {
  size_t rowNumber = threadIdx.x + blockDim.x * blockIdx.x;
  size_t index = noColumn + rowNumber * n_columns;

  if(index < n_columns * n_rows) {
    d_in[index] = value;
  }
}

void SetColumn(Tensor in_, size_t col, float value) {
  int nRows = in_->shape().elements() / in_->shape()[-1];
  int nColumns = in_->shape()[-1];

  int nBlocks = nRows / 512 + ((nRows % 512 == 0) ? 0 : 1);
  int nThreads = std::min(512, nRows);

  gSetColumn<<<nBlocks, nThreads>>>(in_->data(), nColumns, nRows, col, value);
}

void suppressWord(Expr probs, Word id) {
  SetColumn(probs->val(), id, std::numeric_limits<float>::lowest());
}
}  // namespace gpu
}  // namespace marian
