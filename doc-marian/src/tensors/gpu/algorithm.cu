#include "hip/hip_runtime.h"
#include "tensors/gpu/algorithm.h"

// clang-format off
#include "tensors/tensor_operators.h"
#include "tensors/gpu/cuda_helpers.h"
// clang-format on

namespace marian {
namespace gpu {

template <typename T>
void copy(Ptr<Backend> backend, const T* begin, const T* end, T* dest) {
  CUDA_CHECK(hipSetDevice(backend->getDeviceId().no));
  CudaCopy(begin, end, dest);
  CUDA_CHECK(hipStreamSynchronize(0));
}

// clang-format off
template void copy<int8_t>(Ptr<Backend>, const int8_t*, const int8_t*, int8_t*);
template void copy<int16_t>(Ptr<Backend>, const int16_t*, const int16_t*, int16_t*);
template void copy<int32_t>(Ptr<Backend>, const int32_t*, const int32_t*, int32_t*);
template void copy<int64_t>(Ptr<Backend>, const int64_t*, const int64_t*, int64_t*);

template void copy<uint8_t>(Ptr<Backend>, const uint8_t*, const uint8_t*, uint8_t*);
template void copy<uint16_t>(Ptr<Backend>, const uint16_t*, const uint16_t*, uint16_t*);
template void copy<uint32_t>(Ptr<Backend>, const uint32_t*, const uint32_t*, uint32_t*);
template void copy<uint64_t>(Ptr<Backend>, const uint64_t*, const uint64_t*, uint64_t*);

template void copy<char>(Ptr<Backend>, const char*, const char*, char*);
template void copy<float>(Ptr<Backend>, const float*, const float*, float*);
template void copy<double>(Ptr<Backend>, const double*, const double*, double*);
// clang-format on

template <typename T>
__global__ void gFill(T* d_in, int size, T val) {
  //auto blocks = gridDim.x;
  auto threadsPerBlock = blockDim.x;
  //for(int bid = 0; bid < size; bid += threadsPerBlock * blocks) {
    int index = /*bid +*/ threadIdx.x + threadsPerBlock * blockIdx.x;
    if(index < size) {
      d_in[index] = val;
    }
  //}
}

template <typename T>
void fill(Ptr<Backend> backend, T* begin, T* end, T value) {
  int size = end - begin;
  if (size == 0)
    return;
  CUDA_CHECK(hipSetDevice(backend->getDeviceId().no));
  int threadsPerBlock = std::min(MAX_THREADS, size);
  int blocks = (size / threadsPerBlock) + (size % threadsPerBlock != 0); // @TODO: (size+threadsPerBlock-1)/threadsPerBlock or CeilDiv(a,b)
  gFill<<<blocks, threadsPerBlock>>>(begin, size, value);
  CUDA_CHECK(hipStreamSynchronize(0));
}

template void fill<bool>(Ptr<Backend>, bool*, bool*, bool);
template void fill<int8_t>(Ptr<Backend>, int8_t*, int8_t*, int8_t);
template void fill<int16_t>(Ptr<Backend>, int16_t*, int16_t*, int16_t);
template void fill<int32_t>(Ptr<Backend>, int32_t*, int32_t*, int32_t);
template void fill<int64_t>(Ptr<Backend>, int64_t*, int64_t*, int64_t);
template void fill<uint8_t>(Ptr<Backend>, uint8_t*, uint8_t*, uint8_t);
template void fill<uint16_t>(Ptr<Backend>, uint16_t*, uint16_t*, uint16_t);
template void fill<uint32_t>(Ptr<Backend>, uint32_t*, uint32_t*, uint32_t);
template void fill<uint64_t>(Ptr<Backend>, uint64_t*, uint64_t*, uint64_t);

template void fill<float>(Ptr<Backend>, float*, float*, float);
template void fill<double>(Ptr<Backend>, double*, double*, double);

void setSparse(Ptr<Backend> backend,
               const std::vector<size_t>& keys,
               const std::vector<float>& values,
               float* data) {
  CUDA_CHECK(hipSetDevice(backend->getDeviceId().no));
  ABORT("no SetSparse");
  // gpu::SetSparse(data, keys, values);
  CUDA_CHECK(hipStreamSynchronize(0));
}

template <typename T>
__global__ void gSwap(T* d_v1, T* d_v2, int size) {
  auto threadsPerBlock = blockDim.x;
  int index = threadIdx.x + threadsPerBlock * blockIdx.x;
  if(index < size) {
    T temp = d_v1[index];
    d_v1[index] = d_v2[index];
    d_v2[index] = temp;
  }
}

template <typename T>
void swap_ranges(Ptr<Backend> backend, T* begin, T* end, T* dest) {
  int size = end - begin;
  if (size == 0)
    return;

  CUDA_CHECK(hipSetDevice(backend->getDeviceId().no));
  int threadsPerBlock = std::min(MAX_THREADS, size);
  int blocks = (size / threadsPerBlock) + (size % threadsPerBlock != 0); // @TODO: (size+threadsPerBlock-1)/threadsPerBlock or CeilDiv(a,b)
  gSwap<<<blocks, threadsPerBlock>>>(begin, dest, size);
  CUDA_CHECK(hipStreamSynchronize(0));
}

// clang-format off
template void swap_ranges<int8_t>(Ptr<Backend>, int8_t*, int8_t*, int8_t*);
template void swap_ranges<int16_t>(Ptr<Backend>, int16_t*, int16_t*, int16_t*);
template void swap_ranges<int32_t>(Ptr<Backend>, int32_t*, int32_t*, int32_t*);
template void swap_ranges<int64_t>(Ptr<Backend>, int64_t*, int64_t*, int64_t*);

template void swap_ranges<uint8_t>(Ptr<Backend>, uint8_t*, uint8_t*, uint8_t*);
template void swap_ranges<uint16_t>(Ptr<Backend>, uint16_t*, uint16_t*, uint16_t*);
template void swap_ranges<uint32_t>(Ptr<Backend>, uint32_t*, uint32_t*, uint32_t*);
template void swap_ranges<uint64_t>(Ptr<Backend>, uint64_t*, uint64_t*, uint64_t*);

template void swap_ranges<char>(Ptr<Backend>, char*, char*, char*);
template void swap_ranges<float>(Ptr<Backend>, float*, float*, float*);
template void swap_ranges<double>(Ptr<Backend>, double*, double*, double*);
// clang-format on

}  // namespace gpu
}  // namespace marian
