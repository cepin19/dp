#include <hip/hip_runtime.h>
#include <iostream>

#include "tensors/device.h"
#include "tensors/gpu/cuda_helpers.h"

namespace marian {
namespace gpu {

Device::~Device() {
  // Note: The CUDA_CHECKs here are not throwing, but will terminate the program.
  CUDA_CHECK(hipSetDevice(deviceId_.no));
  if(data_) {
    CUDA_CHECK(hipFree(data_));
  }
  CUDA_CHECK(hipDeviceSynchronize());
}

void Device::reserve(size_t size) {
  size = align(size);
  CUDA_CHECK(hipSetDevice(deviceId_.no));

  ABORT_IF(size < size_ || size == 0,
           "New size must be larger than old size and larger than 0");

  if(data_) {
    // Allocate memory while temporarily parking original content in host memory
    std::vector<uint8_t> temp(size_);
    CUDA_CHECK(hipMemcpy(temp.data(), data_, size_, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(data_));
    LOG(debug, "[memory] Re-allocating from {} to {} bytes on device {}", size_, size, deviceId_.no);
    CUDA_CHECK(hipMalloc(&data_, size));
    CUDA_CHECK(hipMemcpy(data_, temp.data(), size_, hipMemcpyHostToDevice));
    //logCallStack(0);
  } else {
    // No data_ yet: Just alloc.
    LOG(debug, "[memory] Allocating {} bytes in device {}", size, deviceId_.no);
    CUDA_CHECK(hipMalloc(&data_, size));
  }

  size_ = size;
}
}  // namespace gpu
}  // namespace marian
