#include "hip/hip_runtime.h"
#include "tensors/gpu/add.h"

#include "tensors/gpu/cuda_helpers.h"

#include "functional/functional.h"
#include "functional/shape.h"
#include "functional/tensor.h"
#include "functional/tmp.h"

namespace marian {

namespace gpu {

template <size_t K, class Functor, class AggFunctor>
__global__ void gAggregateGeneric(Functor functor, float aggInit, AggFunctor aggFunctor,
                            const functional::Shape full,
                            functional::Tensor<float> out,
                            functional::Array<functional::Tensor<float>, K> ins,
                            float scale = 1.0) {
  int outLength = out.shape().elements();
  bool same = outLength == full.elements();
  for(int i = 0; i < K; ++i)
    same = same && outLength == ins[i].shape().elements();

  constexpr size_t N = functional::Shape::size();
  functional::Array<int, N> len;
  for(int i = 0; i < N; ++i)
    len[i] = full[i] / out.shape()[i];

  functional::Array<int, N> dims;
  for(int bid = 0; bid < outLength; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < outLength) {
      if(same) {
        out[index] = aggFunctor(out[index], functional::apply(functor, ins, index) * scale);
      } else {
        out.shape().dims(index, dims);
        out[index] = aggFunctor(out[index], functional::loops(functor, aggInit, aggFunctor, ins, len, dims) * scale);
      }
    }
  }
}

template <size_t K, class Functor, class AggFunctor>
__global__ void gAggregateEqual(Functor functor, AggFunctor aggFunctor,
                          functional::Tensor<float> out,
                          functional::Array<functional::Tensor<float>, K> ins,
                          float scale,
                          bool broadcast) {
  int length = out.shape().elements();
  functional::Array<int, functional::Shape::size()> dims;

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      functional::Array<int, K> indices;
      indices.fill(index);

      if(broadcast) {
        out.shape().dims(index, dims);
        for(size_t i = 0; i < K; ++i)
          indices[i] = ins[i].shape().bindex(dims);
      }

      out[index] = aggFunctor(out[index], functional::apply(functor, ins, indices) * scale);
    }
  }
}

template <size_t K, class Functor, class AggFunctor>
__global__ void gAggregateReduce(Functor functor, float aggInit, AggFunctor aggFunctor,
                           const functional::Shape full,
                           functional::Tensor<float> out,
                           functional::Array<functional::Tensor<float>, K> ins,
                           float scale = 1.0) {
  int rows = full.elements() / full.back();
  int cols = full.back();

  bool same = true;
  for(int i = 0; i < K; ++i)
    same = same && ins[i].shape().elements() == full.elements();

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share;

      if(same) {
        _sum[threadIdx.x] = aggInit;
        for(int tid = 0; tid < cols; tid += blockDim.x) {
          int id = tid + threadIdx.x;
          if(id < cols)
            _sum[threadIdx.x] = aggFunctor(_sum[threadIdx.x], functional::apply(functor, ins, j * cols + id));
        }
      } else {
        functional::Array<int, functional::Shape::size()> dims;
        _sum[threadIdx.x] = aggInit;

        for(int tid = 0; tid < cols; tid += blockDim.x) {
          int id = tid + threadIdx.x;
          if(id < cols) {
            full.dims(j * cols + id, dims);
            functional::Array<int, K> indices;
            for(int i = 0; i < K; ++i)
              indices[i] = ins[i].shape().bindex(dims);
            _sum[threadIdx.x] = aggFunctor(_sum[threadIdx.x], functional::apply(functor, ins, indices));
          }
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          _sum[threadIdx.x] = aggFunctor(_sum[threadIdx.x], _sum[threadIdx.x + skip]);
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      out[j] = aggFunctor(out[j], _sum[0] * scale);
    }
    __syncthreads();
  }
}

template <class Functor, class AggFunctor, class... Tensors>
void Aggregate(Functor functor, float aggInit, AggFunctor aggFunctor, float scale, marian::Tensor out, Tensors... tensors) {
  hipSetDevice(out->getDeviceId().no);

  auto full = marian::Shape::broadcast({out, tensors...});

  int length = out->shape().elements();

  constexpr size_t K = sizeof...(Tensors);

  functional::Tensor<float> gOut = out;
  functional::Array<functional::Tensor<float>, K> gIns = {tensors...};

  if(full.back() != 1 && out->shape().back() == 1) {
    size_t m = full.elements() / length;
    size_t k = full.back();

    int blocks = std::min(MAX_BLOCKS, (int)m);
    int threads = std::min(MAX_THREADS, (int)k);
    int shared = sizeof(float) * threads;

    gAggregateReduce<<<blocks, threads, shared>>>(functor, aggInit, aggFunctor, full, gOut, gIns, scale);

  } else if(out->shape() == full) {
    int threads = std::min(MAX_THREADS, length);
    int blocks
        = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

    bool broadcast = false;
    for(int i = 0; i < K; ++i)
      broadcast = broadcast || gOut.shape() != gIns[i].shape();
    gAggregateEqual<<<blocks, threads>>>(functor, aggFunctor, gOut, gIns, scale, broadcast);
  } else {
    int threads = std::min(MAX_THREADS, length);
    int blocks
        = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

    gAggregateGeneric<<<blocks, threads>>>(functor, aggInit, aggFunctor, full, gOut, gIns, scale);
  }
}

// @TODO: this is a duplicate; can be removed, but need to redo all the add.inc entries...
template <class Functor, class... Tensors>
void Add(Functor functor, float scale, marian::Tensor out, Tensors... tensors) {
  hipSetDevice(out->getDeviceId().no);

  auto full = marian::Shape::broadcast({out, tensors...});

  int length = out->shape().elements();

  constexpr size_t K = sizeof...(Tensors);

  functional::Tensor<float> gOut = out;
  functional::Array<functional::Tensor<float>, K> gIns = {tensors...};

  auto addFunctor = functional::_1 + functional::_2;

  if(full.back() != 1 && out->shape().back() == 1) {
    size_t m = full.elements() / length;
    size_t k = full.back();

    int blocks = std::min(MAX_BLOCKS, (int)m);
    int threads = std::min(MAX_THREADS, (int)k);
    int shared = sizeof(float) * threads;

    gAggregateReduce<<<blocks, threads, shared>>>(functor, 0, addFunctor, full, gOut, gIns, scale);

  } else if(out->shape() == full) {
    int threads = std::min(MAX_THREADS, length);
    int blocks
        = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

    bool broadcast = false;
    for(int i = 0; i < K; ++i)
      broadcast = broadcast || gOut.shape() != gIns[i].shape();
    gAggregateEqual<<<blocks, threads>>>(functor, addFunctor, gOut, gIns, scale, broadcast);
  } else {
    int threads = std::min(MAX_THREADS, length);
    int blocks
        = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

    gAggregateGeneric<<<blocks, threads>>>(functor, 0, addFunctor, full, gOut, gIns, scale);
  }
}

#include "tensors/gpu/add.inc"
}  // namespace gpu
}  // namespace marian
